#include <iostream>
#include "hip/hip_runtime.h"  

// 矩阵乘法:行共享储存矩阵乘法
// m*l l*n
__global__ void matrix_mul(float* x, float * y, float* z, int m, int n, int l)
{
  int bidx = blockIdx.x;
  int tidx = threadIdx.x;
  extern __shared__ float data[];
  for(int i = tidx; i < l; i += blockDim.x) {
    data[i] = x[bidx*l + i];
  }

  // 注意调用这个函数保证该 block 里面所有的线程同步， 
  // 因为该 block 里面所有的线程需要协同工作，一起将 m*l 矩阵中的第 bidx 行的元素写入 data 中。
  __syncthreads();

  // for(; bidx < m; bidx += gridDim.x)
  {
    for(;tidx < n; tidx += blockDim.x) {
      for(int i = 0; i < l; i++) {
        z[bidx*n + tidx] += data[i] * y[i*n + tidx];
      }
    }
  }
}

int main()
{
  int M = 2048;
  int L = 1024;
  int N = 512;

  // 申请host内存
  float *x = NULL;
  float *y = NULL;
  float *z = NULL;
  x = (float*)malloc(M*L*sizeof(float));
  y = (float*)malloc(L*N*sizeof(float));
  z = (float*)malloc(M*N*sizeof(float));

  if(x == NULL || y == NULL || z == NULL)
    return 0;
  
  // 初始化数据
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < L; ++j) {
      x[i*L + j] = 1.1;
    }
  }
  for (int i = 0; i < L; ++i) {
    for (int j = 0; j < N; ++j) {
      y[i*N + j] = 1.1;
    }
  }
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      z[i*N + j] = 0;
    }
  }

  // 申请device内存
  float *d_x, *d_y, *d_z;
  hipMalloc((void**)&d_x, M*L*sizeof(float));
  hipMalloc((void**)&d_y, L*N*sizeof(float));
  hipMalloc((void**)&d_z, M*N*sizeof(float));

  // 将host数据拷贝到device
  hipMemcpy((void*)d_x, (void*)x, M*L*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy((void*)d_y, (void*)y, L*N*sizeof(float), hipMemcpyHostToDevice);

  // 定义kernel的执行配置
  dim3 blocks(M);
  dim3 threads(1024);
  matrix_mul <<<blocks, threads, sizeof(float)*L>>>(d_x, d_y, d_z, M, N, L);

  // 将device得到的结果拷贝到host
  hipMemcpy((void*)z, (void*)d_z, M*N*sizeof(float), hipMemcpyDeviceToHost);

  // 输出前10个数值
  for(int i = 0; i < 10; i++) {
    std::cout << z[i] << " ";
  }
  std::cout << std::endl;

  std::cout << "Done!" << std::endl;

  // 释放device内存
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  // 释放host内存
  free(x);
  free(y);
  free(z);

  return 0;
}
